#include "hip/hip_runtime.h"
/***********************************************************************************
  Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

  Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Created by Pawan Harish.

  Changes for sharedalloc implementation by Mohammad Dashti
 ************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <unistd.h>
#include <sys/syscall.h>
#define gpu_hook(x) syscall(380,x)

static void HandleError( hipError_t err, const char *file, int line ) {
    
    if (err != hipSuccess) {
        
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    
    }

}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}


#define MAX_THREADS_PER_BLOCK 512

int no_of_nodes;
int edge_list_size;
FILE *fp;

//Structure to hold a node information
struct Node
{
	int starting;
	int no_of_edges;
};

#include "kernel.cu"
#include "kernel2.cu"

void BFSGraph(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
   hipFree(0); //setup context to be able to gpu_hook
   gpu_hook(1);
   gpu_hook(3);
	no_of_nodes=0;
	edge_list_size=0;
	BFSGraph( argc, argv);
}

void Usage(int argc, char**argv){

fprintf(stderr,"Usage: %s <input_file>\n", argv[0]);

}
////////////////////////////////////////////////////////////////////////////////
//Apply BFS on a Graph using CUDA
////////////////////////////////////////////////////////////////////////////////
void BFSGraph( int argc, char** argv) 
{

    char *input_f;
	if(argc!=2){
	Usage(argc, argv);
	exit(0);
	}
	
	input_f = argv[1];
	printf("Reading File\n");
	//Read in Graph from a file
	fp = fopen(input_f,"r");
	if(!fp)
	{
		printf("Error Reading graph file\n");
		return;
	}

	int source = 0;

	fscanf(fp,"%d",&no_of_nodes);

	int num_of_blocks = 1;
	int num_of_threads_per_block = no_of_nodes;

	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(no_of_nodes>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}
   Node* d_graph_nodes;
	HANDLE_ERROR(hipHostAlloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes,0)) ;

   //Copy the Mask to device memory
	bool* d_graph_mask;
	HANDLE_ERROR(hipHostAlloc( (void**) &d_graph_mask, sizeof(bool)*no_of_nodes,0)) ;

	bool* d_updating_graph_mask;
	HANDLE_ERROR(hipHostAlloc( (void**) &d_updating_graph_mask, sizeof(bool)*no_of_nodes,0)) ;

	//Copy the Visited nodes array to device memory
	bool* d_graph_visited;
	HANDLE_ERROR(hipHostAlloc( (void**) &d_graph_visited, sizeof(bool)*no_of_nodes,0)) ;


	int start, edgeno;   
	// initalize the memory
	for( unsigned int i = 0; i < no_of_nodes; i++) 
	{
		fscanf(fp,"%d %d",&start,&edgeno);
		d_graph_nodes[i].starting = start;
		d_graph_nodes[i].no_of_edges = edgeno;
		d_graph_mask[i]=false;
		d_updating_graph_mask[i]=false;
		d_graph_visited[i]=false;
	}

	//read the source node from the file
	fscanf(fp,"%d",&source);
	source=0;

	//set the source node as true in the mask
	d_graph_mask[source]=true;
	d_graph_visited[source]=true;

	fscanf(fp,"%d",&edge_list_size);
   //Copy the Edge List to device Memory
	int *d_graph_edges;
	HANDLE_ERROR(hipHostAlloc( (void**) &d_graph_edges, sizeof(int)*edge_list_size,0)) ;

	int id,cost;
	for(int i=0; i < edge_list_size ; i++)
	{
		fscanf(fp,"%d",&id);
		fscanf(fp,"%d",&cost);
		d_graph_edges[i] = id;
	}

	if(fp)
		fclose(fp);    

	printf("Read File\n");

	// allocate device memory for result
	int* d_cost;
	HANDLE_ERROR(hipHostAlloc( (void**) &d_cost, sizeof(int)*no_of_nodes,0));

   for(int i=0;i<no_of_nodes;i++)
		d_cost[i]=-1;
	d_cost[source]=0;
	
	//make a bool to check if the execution is over
	bool *stop;
	HANDLE_ERROR(hipHostAlloc( (void**) &stop, sizeof(bool),0));

	printf("Copied Everything to GPU memory\n");

	// setup execution parameters
	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	int k=0;
	printf("Start traversing the tree\n");
	//Call the Kernel untill all the elements of Frontier are not false
	do
	{
		//if no thread changes this value then the loop stops
		*stop=false;
      gpu_hook(2);
		Kernel<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges, d_graph_mask, d_updating_graph_mask, d_graph_visited, d_cost, no_of_nodes);
		// check if kernel execution generated and error
		
      hipDeviceSynchronize();
      gpu_hook(5);

		Kernel2<<< grid, threads, 0 >>>( d_graph_mask, d_updating_graph_mask, d_graph_visited, stop, no_of_nodes);
		// check if kernel execution generated and error
		

      hipDeviceSynchronize();
      gpu_hook(5);
		k++;
	}
	while(*stop);
   HANDLE_ERROR(hipGetLastError());


	printf("Kernel Executed %d times\n",k);


	//Store the result into a file
	FILE *fpo = fopen("result.txt","w");
	for(int i=0;i<no_of_nodes;i++)
		fprintf(fpo,"%d) cost:%d\n",i,d_cost[i]);
	fclose(fpo);
	printf("Result stored in result.txt\n");


	hipHostFree(d_graph_nodes);
	hipHostFree(d_graph_edges);
	hipHostFree(d_graph_mask);
	hipHostFree(d_updating_graph_mask);
	hipHostFree(d_graph_visited);
	hipHostFree(d_cost);
}
